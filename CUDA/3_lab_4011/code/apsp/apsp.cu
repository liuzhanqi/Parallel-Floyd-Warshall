#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

__global__ void testKernel(float *g_idata, float *g_odata) {
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

void par_apsp(int N, float *mat) {
	unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = mat;

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
        h_idata[i] = (float)i;

    // allocate device memory
    float *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

    // allocate device memory for result
    float *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads, hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // compute reference solution
    float *reference = (float *) malloc(mem_size);
    computeGold(reference, h_idata, num_threads);

    // custom output handling when no regression test running
    // in this case check if the result is equivalent to the expected solution
    bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    if (bTestResult)
    	printf("SUCCESS\n");
    else
    	printf("FAILED - RESULT WRONG\n");
}
